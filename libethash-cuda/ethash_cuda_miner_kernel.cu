#include "hip/hip_runtime.h"
/*
* Genoil's CUDA mining kernel for Ethereum
* based on Tim Hughes' opencl kernel.
* thanks to sp_, trpuvot, djm34, cbuchner for things i took from ccminer.
*/

#include "ethash_cuda_miner_kernel.h"
#include "ethash_cuda_miner_kernel_globals.h"
#include "cuda_helper.h"

#include "fnv.cuh"

#define copy(dst, src, count) for (int i = 0; i != count; ++i) { (dst)[i] = (src)[i]; }


#if __CUDA_ARCH__ < SHUFFLE_MIN_VER
#include "keccak_u64.cuh"
#include "dagger_shared.cuh"
#else
#include "keccak.cuh"
#include "dagger_shuffled.cuh"
#endif

template <uint32_t _PARALLEL_HASH>
__global__ void 
ethash_search(
	volatile uint32_t* g_output,
	uint64_t start_nonce
	)
{
	uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;	
        uint64_t hash = compute_hash<_PARALLEL_HASH>(start_nonce + gid);
	if (cuda_swab64(hash) > d_target) return;
	uint32_t index = atomicInc(const_cast<uint32_t*>(g_output), SEARCH_RESULT_BUFFER_SIZE - 1) + 1;
	g_output[index] = gid;
}

void run_ethash_search(
	uint32_t blocks,
	uint32_t threads,
	uint32_t sharedbytes,
	hipStream_t stream,
	volatile uint32_t* g_output,
	uint64_t start_nonce,
	uint32_t parallelHash
)
{
	switch (parallelHash)
	{
		case 1: ethash_search <1> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		case 2: ethash_search <2> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		case 3: ethash_search <3> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		case 4: ethash_search <4> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		case 5: ethash_search <5> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		case 6: ethash_search <6> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		case 7: ethash_search <7> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		case 8: ethash_search <8> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
		default: ethash_search <4> <<<blocks, threads, sharedbytes, stream >>>(g_output, start_nonce); break;
	}
	CUDA_SAFE_CALL(hipGetLastError());
}

#define ETHASH_DATASET_PARENTS 256
#define NODE_WORDS (64/4)


__global__ void
ethash_calculate_dag_item(uint32_t start)
{
	uint32_t const node_index = start + blockIdx.x * blockDim.x + threadIdx.x;
	if (node_index > d_dag_size * 2) return;

	hash200_t dag_node;
	copy(dag_node.uint4s, d_light[node_index % d_light_size].uint4s, 4);
	dag_node.words[0] ^= node_index;
	SHA3_512(dag_node.uint2s);

	const int thread_id = threadIdx.x & 3;

	for (uint32_t i = 0; i != ETHASH_DATASET_PARENTS; ++i) {
		uint32_t parent_index = fnv(node_index ^ i, dag_node.words[i % NODE_WORDS]) % d_light_size;
#if __CUDA_ARCH__ < SHUFFLE_MIN_VER
		for (unsigned w = 0; w != 4; ++w) {
			dag_node.uint4s[w] = fnv4(dag_node.uint4s[w], d_light[parent_index].uint4s[w]);
		}
#else
		for (uint32_t t = 0; t < 4; t++) {

#if CUDA_VERSION < SHUFFLE_DEPRECATED
			uint32_t shuffle_index = __shfl(parent_index, t, 4);
#else
			uint32_t shuffle_index = __shfl_sync(0xFFFFFFFF,parent_index, t, 4);
#endif

			uint4 p4 = d_light[shuffle_index].uint4s[thread_id];
			for (int w = 0; w < 4; w++) {

#if CUDA_VERSION < SHUFFLE_DEPRECATED
				uint4 s4 = make_uint4(__shfl(p4.x, w, 4), __shfl(p4.y, w, 4), __shfl(p4.z, w, 4), __shfl(p4.w, w, 4));
#else
				uint4 s4 = make_uint4(__shfl_sync(0xFFFFFFFF,p4.x, w, 4), __shfl_sync(0xFFFFFFFF,p4.y, w, 4), __shfl_sync(0xFFFFFFFF,p4.z, w, 4), __shfl_sync(0xFFFFFFFF,p4.w, w, 4));
#endif
				if (t == thread_id) {
					dag_node.uint4s[w] = fnv4(dag_node.uint4s[w], s4);
				}
			}
		}


#endif		
	}
	SHA3_512(dag_node.uint2s);
	hash64_t * dag_nodes = (hash64_t *)d_dag;

#if __CUDA_ARCH__ < SHUFFLE_MIN_VER
	for (uint32_t i = 0; i < 4; i++) {
		dag_nodes[node_index].uint4s[i] =  dag_node.uint4s[i];
	}
#else
	for (uint32_t t = 0; t < 4; t++) {
#if CUDA_VERSION < SHUFFLE_DEPRECATED
		uint32_t shuffle_index = __shfl(node_index, t, 4);
#else
		uint32_t shuffle_index = __shfl_sync(0xFFFFFFFF,node_index, t, 4);
#endif
		uint4 s[4];
		for (uint32_t w = 0; w < 4; w++) {
#if CUDA_VERSION < SHUFFLE_DEPRECATED
			s[w] = make_uint4(__shfl(dag_node.uint4s[w].x, t, 4), __shfl(dag_node.uint4s[w].y, t, 4), __shfl(dag_node.uint4s[w].z, t, 4), __shfl(dag_node.uint4s[w].w, t, 4));
#else
			s[w] = make_uint4(__shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].x, t, 4), __shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].y, t, 4), __shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].z, t, 4), __shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].w, t, 4));
#endif
		}
		dag_nodes[shuffle_index].uint4s[thread_id] = s[thread_id];
	}
#endif		 
}

void ethash_generate_dag(
	uint64_t dag_size,
	uint32_t blocks,
	uint32_t threads,
	hipStream_t stream,
	int device
	)
{
	uint32_t const work = (uint32_t)(dag_size / sizeof(hash64_t));

	uint32_t fullRuns = work / (blocks * threads);
	uint32_t const restWork = work % (blocks * threads);
	if (restWork > 0) fullRuns++;
	for (uint32_t i = 0; i < fullRuns; i++)
	{
		ethash_calculate_dag_item <<<blocks, threads, 0, stream >>>(i * blocks * threads);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		printf("CUDA#%d: %.0f%%\n",device, 100.0f * (float)i / (float)fullRuns);
	}
	//printf("GPU#%d 100%%\n");
	CUDA_SAFE_CALL(hipGetLastError());
}

void set_constants(
	hash128_t* _dag,
	uint32_t _dag_size,
	hash64_t * _light,
	uint32_t _light_size
	)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_dag), &_dag, sizeof(hash128_t *)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), &_dag_size, sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_light), &_light, sizeof(hash64_t *)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_light_size), &_light_size, sizeof(uint32_t)));
}

void set_header(
	hash32_t _header
	)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_header), &_header, sizeof(hash32_t)));
}

void set_target(
	uint64_t _target
	)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_target), &_target, sizeof(uint64_t)));
}
